#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

const char * datafilename = "data/channel256_out";
const char * spectrumfilename = "data/channel256_spectrum";
const int fftlen = 2048;

int main()
{

    FILE *datafile = fopen(datafilename,"r");
    int next_real,next_imaginary; 
    hipfftComplex *roachdata;
    hipfftComplex *gpudata;
    hipfftComplex *gpuspectrum;
    hipfftComplex *cpuspectrum;
    
    hipHostMalloc(&roachdata, sizeof(hipfftComplex)*fftlen, hipHostMallocDefault);
    hipMalloc(&gpudata, sizeof(hipfftComplex)*fftlen);
    hipMalloc(&gpuspectrum, sizeof(hipfftComplex)*fftlen);
    hipHostMalloc(&cpuspectrum, sizeof(hipfftComplex)*fftlen, hipHostMallocDefault);
    
    for(int i=0; i<fftlen && fscanf(datafile, "%d %d\n", &next_real, &next_imaginary) != EOF;i++)
    {
        roachdata[i].x = next_real;
        roachdata[i].y = next_imaginary;
    }
    
    
    // allocate device memory for the fft
    hipMalloc((void**)&gpudata,sizeof(hipfftComplex)*fftlen);
    hipMalloc((void**)&gpuspectrum,sizeof(hipfftComplex)*fftlen);

    static hipfftHandle plan;
    hipfftPlan1d(&plan,fftlen,HIPFFT_C2C, 1);


    // allocate device memory and copy over data
    hipMemcpy(gpudata, roachdata, sizeof(hipfftComplex)*fftlen, hipMemcpyHostToDevice);
    
    // run the fft
    hipfftExecC2C(plan,gpudata,gpuspectrum,HIPFFT_FORWARD);
    // copy the result back
    hipMemcpy(cpuspectrum, gpuspectrum, sizeof(hipfftComplex)*fftlen, hipMemcpyDeviceToHost);

    FILE *spectrumfile = fopen(spectrumfilename,"w");
    
    for(int i=0; i<fftlen; i++)
    {
        fprintf(spectrumfile, "%f %f\n", cpuspectrum[i].x, cpuspectrum[i].y);
    }
    
    hipfftDestroy(plan);
    hipHostFree(roachdata);
    hipFree(gpudata);
    hipFree(gpuspectrum);
    hipHostFree(cpuspectrum);
}