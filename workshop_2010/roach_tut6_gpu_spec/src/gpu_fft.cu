#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

const char * datafilename = "data/channel256_out";
const char * spectrumfilename = "data/channel256_spectrum";
const int fftlen = 2048;

int main()
{
    //open the file containing our recorded data
    FILE *datafile = fopen(datafilename,"r");
    int next_real,next_imaginary; 
    
    //allocate space for the time domain and ffted data on the cpu and the gpu
    hipfftComplex *roachdata;
    hipfftComplex *gpudata;
    hipfftComplex *gpuspectrum;
    hipfftComplex *cpuspectrum;
    
    hipHostMalloc(&roachdata, sizeof(hipfftComplex)*fftlen, hipHostMallocDefault);
    hipMalloc(&gpudata, sizeof(hipfftComplex)*fftlen);
    hipMalloc(&gpuspectrum, sizeof(hipfftComplex)*fftlen);
    hipHostMalloc(&cpuspectrum, sizeof(hipfftComplex)*fftlen, hipHostMallocDefault);
    
    //read in the time domain data from the file
    for(int i=0; i<fftlen && fscanf(datafile, "%d %d\n", &next_real, &next_imaginary) != EOF;i++)
    {
        roachdata[i].x = next_real;
        roachdata[i].y = next_imaginary;
    }
    
    //create an fft plan
    static hipfftHandle plan;
    hipfftPlan1d(&plan,fftlen,HIPFFT_C2C, 1);


    // allocate device memory and copy over data
    hipMemcpy(gpudata, roachdata, sizeof(hipfftComplex)*fftlen, hipMemcpyHostToDevice);
    
    // run the fft
    hipfftExecC2C(plan,gpudata,gpuspectrum,HIPFFT_FORWARD);
    // copy the result back
    hipMemcpy(cpuspectrum, gpuspectrum, sizeof(hipfftComplex)*fftlen, hipMemcpyDeviceToHost);

    FILE *spectrumfile = fopen(spectrumfilename,"w");
    
    for(int i=0; i<fftlen; i++)
    {
        fprintf(spectrumfile, "%f %f\n", cpuspectrum[i].x, cpuspectrum[i].y);
    }
    
    hipfftDestroy(plan);
    hipHostFree(roachdata);
    hipFree(gpudata);
    hipFree(gpuspectrum);
    hipHostFree(cpuspectrum);
}